#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "lodepng/lodepng.h"
#include "img.h"

#define WIDTH 2048
#define HEIGHT 2048
#define IMG_SIZE (WIDTH*HEIGHT)

using namespace std;

void readFile(const char* address, float* data, float norm = 1)
{
  unsigned char *image;
  uint i_w, i_h;
  if(lodepng_decode32_file(&image,&i_w,&i_h,address))
  {
    cout << "Erreur lors de l'ouverture du fichier." << endl;
    exit(-1);
  }
  if(i_w != WIDTH || i_h != HEIGHT)
  {
    cout << "Taille de l'image incorecte: (" << i_w << ", " << i_h << ") au lieu de (" << WIDTH << ", " << HEIGHT << ")." << endl;
    exit(-1);
  }
  for(int i = 0; i < IMG_SIZE; i++)
  {
    data[i] = image[4*i]/3.f+image[4*i+1]/3.f+image[4*i+2]/3.f;
  }
  free(image);
}

typedef unsigned int uint;

int main(int argc, char** argv)
{
  uint w = WIDTH;
  uint h = HEIGHT;
  float *tab = new float [w*h];
  readFile("lena.png", tab);

  float *devTab;
  hipMalloc(&devTab,w*h*sizeof(float));
  hipMemcpy(devTab,tab,w*h*sizeof(float),hipMemcpyHostToDevice);

  delete tab;

  Image img(w,h,devTab);

  Image tile = img.makeTile(10,10,1024,1024);
  Image ttile = tile.makeTile(10,10,512,512);

  cout << "Image: " << img.getVal(500,500) << endl;
  cout << "Tuile: " << tile.getVal(490,490) << endl;
  cout << "Tuile de tuile: " << ttile.getVal(480,480) << endl;

  tile.writeToFile("tile.png");
  ttile.writeToFile("ttile.png");
  return EXIT_SUCCESS;
}
