#include "hip/hip_runtime.h"
#include <iostream>
/*#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>*/

#include "lodepng/lodepng.h"
#include "img.h"


using namespace std;

void readFile(const char* address, float* data, float norm = 1)
{
  unsigned char *image;
  uint i_w, i_h;
  if(lodepng_decode32_file(&image,&i_w,&i_h,address))
  {
    cout << "Erreur lors de l'ouverture du fichier." << endl;
    exit(-1);
  }
  cout << "Dimensions de l'image: " << i_w << "x" << i_h << endl;
  for(int i = 0; i < i_w*i_h; i++)
  {
    data[i] = image[4*i]/3.f+image[4*i+1]/3.f+image[4*i+2]/3.f;
  }
  free(image);
}

typedef unsigned int uint;

int main(int argc, char** argv)
{
  uint w = 2048;
  uint h = 2048;
  float *tab = new float [w*h];
  readFile("lena.png", tab);
  srand(time(NULL));

  float *devTab;
  hipMalloc(&devTab,w*h*sizeof(float));
  hipMemcpy(devTab,tab,w*h*sizeof(float),hipMemcpyHostToDevice);

  delete tab;

  Image img(w,h,devTab);

  Image tile = img.makeTile(10,10,1024,1024);
  Image ttile = tile.makeTile(10,10,512,512);

  uint x = (int)(rand()%500+20);
  uint y = (int)(rand()%500+20);

  cout << "Point: " << x << ", " << y << endl;
  cout << "Image: " << img.getVal(x,y) << endl;
  cout << "Tuile: " << tile.getVal(x-10,y-10) << endl;
  cout << "sous-tuile: " << ttile.getVal(x-20,y-20) << endl;
  //tile.writeToFile("tile.png");

  float *devOut;
  hipMalloc(&devOut, w*h*sizeof(float));
  float2 *devU;
  hipMalloc(&devU, w*h*sizeof(float2));

  uint tile_w = 1024;
  uint tile_h = 1024;

  float2 *U = new float2 [tile_w*tile_h];
  for(int i = 0; i < tile_w; i++)
  {
    for(int j = 0; j < tile_h; j++)
    {
      U[i+tile_w*j] = make_float2(2.f*i/tile_w-1.f,0.f);
    }
  }
  
  hipMemcpy(devU,U,tile_w*tile_h*sizeof(float2),hipMemcpyHostToDevice);
  float k = 15.f;
  
  float2 *devDisp;
  hipMalloc(&devDisp, tile_w*tile_h*sizeof(float2));

  makeDisplacement(devDisp,k,devU,tile_w,tile_h);
  tile.interpLinear(devOut,devDisp,tile_w*tile_h);
  hipDeviceSynchronize();
  Image out(tile_w,tile_h,devOut);
  out.writeToFile("out.png");


  hipFree(devOut);
  hipFree(devU);
  delete U;


  
  return EXIT_SUCCESS;
}
