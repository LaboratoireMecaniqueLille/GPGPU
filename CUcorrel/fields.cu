#include "hip/hip_runtime.h"
#include "CUcorrel.h"

void writeFields(float2* devFields)
{
  //Assignation des champs
  size_t taille2 = WIDTH*HEIGHT*sizeof(float2);
  float2 *field = (float2*)malloc(PARAMETERS*taille2);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 1; // Move X
      field[i+WIDTH*j].y = 0;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Move Y
      field[i+WIDTH*j].y = 1;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 1.4142135624*(j-HEIGHT/2.)/HEIGHT; // Rotation
      field[i+WIDTH*j].y = 1.4142135624*(WIDTH/2.-i)/WIDTH;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+2*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = (float)i/WIDTH; // Stretch X
      field[i+WIDTH*j].y = 0;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+3*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Stretch Y
      field[i+WIDTH*j].y = (float)j/HEIGHT;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+4*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = (float)j/HEIGHT; // Shear X
      field[i+WIDTH*j].y = 0; 
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+5*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Shear Y
      field[i+WIDTH*j].y = (float)i/WIDTH;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+6*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
}
