#include "hip/hip_runtime.h"
#include "CUcorrel.h"
//#include "util.h"
//#include <iostream>

//using namespace std;

//6 champs: 3 mvt de corps solide, 2 déformations uniformes et 1 cisaillement

void writeFields(float2* devFields, uint w, uint h)
{
//  cout << "W: " << w << "\nH: " << h << endl;
  //Assignation des champs
  size_t taille2 = w*h*sizeof(float2);
  float2 *field = (float2*)malloc(taille2);

  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 1.f; // Move X
      field[i+w*j].y = 0.f;
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 0.f; // Move Y
      field[i+w*j].y = 1.f;
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields+w*h,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 1.4142135624f*(j-h/2.f)/h; // Rotation
      field[i+w*j].y = 1.4142135624f*(w/2.f-i)/w;
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields+2*w*h,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 2.f*i/w-1.f; // Stretch X
      field[i+w*j].y = 0.f;
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields+3*w*h,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 0.f; // Stretch Y
      field[i+w*j].y = 2.f*j/h-1.f;
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields+4*w*h,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < w; i++)
  {
    for(int j = 0; j < h; j++)
    {
      field[i+w*j].x = 1.4142135624f*((float)j/h-.5f); // Shear
      field[i+w*j].y = 1.4142135624f*((float)i/w-.5f); 
    }
  }
  //printMat2D(field,w,h,w/16);
  hipMemcpy(devFields+5*w*h,field,taille2,hipMemcpyHostToDevice);
}
