#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "kernels.cuh"
#include "CUcorrel.h"
#include "util.h"

using namespace std;


int main(int argc, char** argv)
{

  struct timeval t0, t1, t2; // Pour mesurer les durées d'exécution
  size_t taille = WIDTH*HEIGHT*sizeof(float); // Taille d'un tableau contenant une image
  size_t taille2 = WIDTH*HEIGHT*sizeof(float2); // idem à 2 dimensions (fields)
  int nbIter=20; // Le nombre d'itérations
  char iAddr[10] = "img.csv"; // Le nom du fichier à ouvrir
  float *orig = (float*)malloc(taille); // le tableau contenant l'image sur l'hôte
  dim3 blocksize(min(32,WIDTH),min(32,HEIGHT)); // Pour l'appel aux kernels sur toute l'image
  dim3 gridsize((WIDTH+31)/32,(HEIGHT+31)/32); // ...
  dim3 tailleMat(PARAMETERS,PARAMETERS); // La taille de la hessienne

  float *devOrig; // Image originale
  float *devGradX; // Gradient de l'image d'origine par rapport à X
  float *devGradY; // .. à Y
  float2 *devFields; // Contient les PARAMETERS champs de déplacements élémentaires à la suite dont on cherche l'influence par autant de paramètres
  float *devG; // Les PARAMETERS matrices gradient*champ
  float *devParam; // Contient la valeur actuelle calculée des paramètres
  float *devDef; // Image déformée à recaler (ici calculée à partir de l'image d'origine)
  float *devOut; // L'image interpolée à chaque itération
  float *devMatrix; // La hessienne utilisée pour la méthode de Newton
  float *devInv;  // L'inverse de la Hessienne
  float *devVec; // Vecteur pour stocker les PARAMETERS valeurs du gradient à chaque itération
  float *devVecStep; // Multiplie terme à terme la direction avant le l'ajouter aux paramètres

  srand(time(NULL)); // Seed pour générer le bruit avec rand()

  // ---------- Allocation tous les tableaux du device ---------
  hipMalloc(&devOrig,taille);
  hipMalloc(&devGradX,taille);
  hipMalloc(&devGradY,taille);
  hipMalloc(&devFields,PARAMETERS*taille2);
  hipMalloc(&devG,PARAMETERS*taille);
  hipMalloc(&devParam,PARAMETERS*sizeof(float));
  hipMalloc(&devDef,taille);
  hipMalloc(&devOut,taille);
  hipMalloc(&devMatrix,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devInv,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devVec,PARAMETERS*sizeof(float));
  hipMalloc(&devVecStep,PARAMETERS*sizeof(float));

  // ---------- Lecture du fichier et écriture sur le device ---------
  readFile(iAddr,orig,256);
  hipMemcpy(devOrig,orig,taille,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  cout << "Image d'origine" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  // --------- Initialisation de la texture et calcul des gradients ---------
  gettimeofday(&t1,NULL);
  initCuda(devOrig);
  gradient<<<gridsize,blocksize>>>(devGradX,devGradY);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "\nCalcul des gradients: " << timeDiff(t1,t2) << " ms." << endl;

  //-------- Affichage des gradients -------
  cout << "Gradient X:" << endl;
  hipMemcpy(orig,devGradX,taille,hipMemcpyDeviceToHost);
  printMat(orig,WIDTH,HEIGHT,256);

  // --------- Écriture des fields définis dans fields.cu ----------
  writeFields(devFields);

  // --------- Calcul des matrices G ----------
  gettimeofday(&t1,NULL);
  makeG<<<1,PARAMETERS>>>(devG,devFields,devGradX,devGradY);
  hipDeviceSynchronize();
  if(hipGetLastError() == hipErrorOutOfMemory)
  {cout << "Erreur d'allocation (manque de mémoire graphique ?)" << endl;exit(-1);}
  else if(hipGetLastError() != hipSuccess)
  {cout << "Erreur lors de l'allocation." << endl;exit(-1);}
  gettimeofday(&t2,NULL);
  cout << "Calcul des matrices G: " << timeDiff(t1,t2) << " ms." << endl;

  // ------- [Facultatif] Écriture des G en .csv pour les visualiser -----------
  /*
  char oAddr[3];
  for(int i = 0;i < PARAMETERS;i++)
  {
  hipMemcpy(orig,devG+i*WIDTH*HEIGHT,taille,hipMemcpyDeviceToHost);
  sprintf(oAddr,"%d",i);
  writeFile(oAddr, orig, 1);
  }
  */
  
  // --------- Allocation et assignation des paramètres de déformation de devDef ----------
  float param[PARAMETERS] = {-.2,-2.318,3.22,-1.145,1.37,2.3,0};
  cout << "Paramètres réels: ";
  for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
  cout << endl;
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Calcul de l'image à recaler ----------
  deform2D<<<gridsize,blocksize>>>(devDef,devFields,devParam);

  // ---------- Bruitage de l'image déformée ---------
  for(int i = 0; i < WIDTH*HEIGHT ; i++)
  { 
    orig[i] = (float)rand()/RAND_MAX*4-2;
  }
  hipMemcpy(devOut,orig,taille,hipMemcpyHostToDevice);// Pour ajouter le bruit
  addVec<<<WIDTH*HEIGHT/1024,1024>>>(devDef,devOut);

  // ---------- Calcul de la Hessienne ----------
  gettimeofday(&t1,NULL);
  makeMatrix<<<1,tailleMat>>>(devMatrix,devG);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Génération de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de la Hessienne ----------
  float test[PARAMETERS*PARAMETERS];
  hipMemcpy(test,devMatrix,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nHessienne:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);

  // ---------- Inversion de la hessienne ----------
  gettimeofday(&t1,NULL);
  invert(devMatrix,devInv);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Inversion de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'inverse ----------
  hipMemcpy(test,devInv,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nMatrice inversée:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);

  // --------- [Facultatif] Écriture de l'image déformée en .csv pour la visualiser ----------
  /*
  char oAddr[10] = "out.csv";
  hipMemcpy(orig,devDef,taille,hipMemcpyDeviceToHost); // Pour récupérer l'image
  writeFile(oAddr, orig, 1);
  */

  // ---------- Écriture des paramètres initiaux ----------
  for(int i = 0; i < PARAMETERS; i++)
  {param[i] = 0;}
  //readParam(argv,param); // Pour tester des valeurs de paramètres par défaut sans recompiler
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Écriture du pas des paramètres ----------
  float vecStep[PARAMETERS] = {2,2,2,2,2,2,2};
  hipMemcpy(devVecStep,vecStep,PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  float res = 10000000000;
  float oldres=0;
  float vec[PARAMETERS];

  for(int i = 0;i < nbIter; i++)
  {
    gettimeofday(&t0,NULL);
    cout << "Boucle n°" << i+1 << endl;
    hipMemcpy(param,devParam,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Paramètres calculés: ";
    for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
    cout << endl;

    gettimeofday(&t1,NULL);
    deform2D<<<gridsize,blocksize>>>(devOut, devFields, devParam);//--
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    cout << "\nInterpolation: " << timeDiff(t1,t2) << "ms." << endl;

    gettimeofday(&t1,NULL);
    gradientDescent(devG, devOut, devDef, devVec);//--
    hipDeviceSynchronize();
    gettimeofday(&t2,NULL);
    cout << "Calcul des gradients des paramètres: " << timeDiff(t1,t2) << " ms." << endl;

    hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Gradient des paramètres:" << endl;
    printMat(vec,PARAMETERS,1);
    
    gettimeofday(&t1,NULL);
    myDot<<<1,PARAMETERS,PARAMETERS*sizeof(float)>>>(devInv,devVec,devVec);//--
    ewMul<<<1,PARAMETERS>>>(devVec,devVecStep);//--
    addVec<<<1,PARAMETERS>>>(devParam,devVec);//--
    hipDeviceSynchronize();
    gettimeofday(&t2,NULL);
    cout << "Mise à jour des valeurs: " << timeDiff(t1,t2) << " ms." << endl;

    hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Direction:" << endl;
    printMat(vec,PARAMETERS,1);

    gettimeofday(&t1, NULL);
    oldres = res;//--
    res = residuals(devOut, devDef, HEIGHT*WIDTH)/HEIGHT/WIDTH;//--
    if(oldres - res < 0)//--
    {cout << "Augmentation de la fonctionnelle !!" << endl;}//--
    gettimeofday(&t2, NULL);
    cout << "\nÉcart: "<< res << ", Calcul de l'écart: " << timeDiff(t1,t2) << "ms." << endl;
    cout << "\nExécution de toute la boucle: " << timeDiff(t0,t2) << "ms.\n**********************\n\n\n" << endl;

  }

  //Vérification d'erreur éventuelle
  hipError_t err;
  err = hipGetLastError();
  cout << "Cuda status: " << ((err == 0)?"OK.":"ERREUR !!") << endl;
  cout << err << endl;
  if(err != 0)
  {cout << hipGetErrorName(err) << endl;}

  //Pour libérer ce qui a été alloué avec initCuda
  cleanCuda();

  //On libère toute la mémoire GPU
  hipFree(devOrig);
  hipFree(devGradX);
  hipFree(devGradY);
  hipFree(devFields);
  hipFree(devG);
  hipFree(devParam);
  hipFree(devDef);
  hipFree(devOut);
  hipFree(devMatrix);
  hipFree(devInv);
  hipFree(devVec);
  hipFree(devVecStep);
  return 0;
}
