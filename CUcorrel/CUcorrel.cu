#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernels.cuh"
#include "CUcorrel.h"
#include "util.h"

using namespace std;


int main(int argc, char** argv)
{
  struct timeval t0, t1, t2, t00; // Pour mesurer les durées d'exécution
  hipError_t err; // Pour récupérer les erreurs éventuelles
  size_t taille = IMG_SIZE*sizeof(float); // Taille d'un tableau contenant une image
  size_t taille2 = IMG_SIZE*sizeof(float2); // idem à 2 dimensions (fields)
  int nbIter=10; // Le nombre d'itérations
  char iAddr[10] = "img.png"; // Le nom du fichier à ouvrir
  float *orig = (float*)malloc(taille); // le tableau contenant l'image sur l'hôte
  dim3 blocksize[LVL]; // Pour l'appel aux kernels sur toute l'image (une pour chaque étage)
  dim3 gridsize[LVL];
  char oAddr[25]; // pour écrire les noms des fichiers de sortie
  float param[PARAMETERS] = {0}; // Stocke les paramètres calculés
  float res; // Le résidu 
  float oldres; // Pour stocker le résidu de l'itération précédente et comparer
  float vec[PARAMETERS]; // Pour stocker sur l'hôte les paramètres calculés
  int c = 0; // Pour compter les boucles et quitter si on ajoute trop
  uint div = 1; // Pour diviser la taille dans les boucles
  for(int i = 0; i < LVL; i++)
  {
    blocksize[i].x = min(32,WIDTH/div);
    blocksize[i].y = min(32,HEIGHT/div);
    blocksize[i].z = 1;
    gridsize[i].x = (WIDTH/div+31)/32;
    gridsize[i].y = (HEIGHT/div+31)/32;
    gridsize[i].z = 1;
    div *= 2;
  }
  dim3 tailleMat(PARAMETERS,PARAMETERS); // La taille de la hessienne

  float *devOrig; // Image originale
  float *devGradX; // Gradient de l'image d'origine par rapport à X
  float *devGradY; // .. à Y
  float2 *devFields[LVL]; // Contient les PARAMETERS champs de déplacements élémentaires à la suite dont on cherche l'influence par autant de paramètres
  float *devParam; // Contient la valeur actuelle calculée des paramètres
  float *devDef[LVL]; // Image déformée à recaler (ici calculée à partir de l'image d'origine)
  float *devOut; // L'image interpolée à chaque itération
  float *devMatrix; // La hessienne utilisée pour la méthode de Newton
  float *devInv;  // L'inverse de la Hessienne
  float *devVec; // Vecteur pour stocker les PARAMETERS valeurs du gradient à chaque itération
  float *devVecStep; // Multiplie terme à terme la direction avant le l'ajouter aux paramètres
  float *devVecOld; // Pour stocker le vecteur précédent et le restaurer si nécessaire

  srand(time(NULL)); // Seed pour générer le bruit avec rand()

  // ---------- Allocation de tous les tableaux du device ---------
  hipMalloc(&devOrig,taille);
  hipMalloc(&devGradX,taille);
  hipMalloc(&devGradY,taille);
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    hipMalloc(&devFields[i],PARAMETERS*taille2/div);
    hipMalloc(&devDef[i],taille/div);
    div*=4;
  }
  hipMalloc(&devParam,PARAMETERS*sizeof(float));
  hipMemcpy(devParam,param,PARAMETERS*sizeof(float),hipMemcpyHostToDevice);
  hipMalloc(&devOut,taille);
  hipMalloc(&devMatrix,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devInv,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devVec,PARAMETERS*sizeof(float));
  hipMalloc(&devVecStep,PARAMETERS*sizeof(float));
  hipMalloc(&devVecOld,PARAMETERS*sizeof(float));
  initCuda();
  if(hipGetLastError() == hipErrorOutOfMemory)
  {cout << "Erreur d'allocation (manque de mémoire graphique ?)" << endl;exit(-1);}
  else if(hipGetLastError() != hipSuccess)
  {cout << "Erreur lors de l'allocation." << endl;exit(-1);}

  // ---------- Écriture des fields définis dans fields.cu ----------
  div = 1;
  for(uint i = 0; i < LVL;i++)
  {
    writeFields(devFields[i],WIDTH/div,HEIGHT/div);
    div *= 2;
  }

  // ---------- Lecture du fichier et écriture sur le device ---------
  readFile(iAddr,orig,256);
  hipMemcpy(devOrig,orig,taille,hipMemcpyHostToDevice);

  // ---------- [Facultatif] Affichage de l'image fixe ----------
  cout << "Image d'origine" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  // ---------- Allocation des bindless textures et copie des données ----------
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipArray *cuArray[LVL];
  hipMallocArray(&cuArray[0], &channelDesc,WIDTH,HEIGHT);
  hipMemcpyToArray(cuArray[0],0,0,orig,IMG_SIZE*sizeof(float),hipMemcpyHostToDevice);

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32;
  resDesc.res.linear.sizeInBytes = IMG_SIZE*sizeof(float);
  resDesc.res.linear.devPtr = cuArray[0];

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  texDesc.addressMode[0] = hipAddressModeBorder; //hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeBorder; //hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t tex[LVL]={0};
  hipCreateTextureObject(&tex[0],&resDesc,&texDesc,NULL);

  div = 2;
  for(int i = 1; i < LVL; i++)
  {
    hipMallocArray(&cuArray[i], &channelDesc,WIDTH/div,HEIGHT/div);
    resDesc.res.linear.sizeInBytes = IMG_SIZE/div/div*sizeof(float);
    resDesc.res.linear.devPtr = cuArray[i];
    genMip(tex[i-1],cuArray[i],WIDTH/div,HEIGHT/div);
    hipCreateTextureObject(&tex[i],&resDesc,&texDesc,NULL);
    div *= 2;
  }

  // --------- [Facultatif] Ecriture de l'image originale aux différentes échelles -------
  /*
  div = 1;
  for(int i=0; i < LVL;i++)
  {
    deform2D<<<gridsize[i],blocksize[i]>>>(tex[i], devOut, devFields[i], devParam, WIDTH/div, HEIGHT/div);
    hipDeviceSynchronize();
    hipMemcpy(orig,devOut,IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    sprintf(oAddr,"out/orig%d.png",i);
    writeFile(oAddr, orig, 0, WIDTH/div, HEIGHT/div);
    div *= 2;
  }
  cout << "OK" << endl;
  */

  // --------- Calcul des matrices G ----------
  gettimeofday(&t1,NULL);
  hipTextureObject_t texG[LVL][PARAMETERS]={{0}};
  hipArray *Garray[LVL][PARAMETERS];
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    gradient<<<gridsize[i],blocksize[i]>>>(tex[i],devGradX,devGradY, WIDTH/div, HEIGHT/div);
    resDesc.res.linear.sizeInBytes = IMG_SIZE/div/div*sizeof(float);
    for(int j = 0; j < PARAMETERS; j++)
    {
      hipMallocArray(&Garray[i][j], &channelDesc, WIDTH/div, HEIGHT/div);
      makeGArray(Garray[i][j],devFields[i]+j*IMG_SIZE/div/div, devGradX, devGradY, WIDTH/div, HEIGHT/div);
      resDesc.res.linear.devPtr = Garray[i][j];
      hipCreateTextureObject(&texG[i][j],&resDesc,&texDesc,NULL);
    }
    div *= 2;
  }
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Calcul des matrices G: " << timeDiff(t1,t2) << " ms." << endl;

  // --------- [Facultatif] Ecriture des G en .png -----------
/*
  hipMemcpy(devParam,param,PARAMETERS*sizeof(float),hipMemcpyHostToDevice);
  div = 1;
  for(int l = 0; l < LVL; l++)
  {
    for(int p = 0; p < PARAMETERS; p++)
    {
      deform2D_b<<<gridsize[l],blocksize[l]>>>(texG[l][p],devOut,devFields[l],devParam,WIDTH/div,HEIGHT/div);
      hipMemcpy(orig,devOut,IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
      sprintf(oAddr,"out/Gl%d-p%d.png",l,p);
      writeFile(oAddr,orig,128,WIDTH/div,HEIGHT/div);
    }
    div*=2;
  }
*/

  // --------- Allocation et assignation des paramètres de déformation de devDef ----------
  float paramI[PARAMETERS] = {-12,30}; // Commenter la boucle pour tester les réglages sur un même jeu de paramètres
  for(int i = 0; i < PARAMETERS; i++)
  paramI[i] = 80.f*rand()/RAND_MAX-40.f;

  cout << "Paramètres réels: ";
  for(int i = 0; i < PARAMETERS;i++){cout << paramI[i] << ", ";}
  cout << endl;
  hipMemcpy(devParam, paramI, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Calcul de l'image à recaler ----------
  deform2D<<<gridsize[0],blocksize[0]>>>(tex[0], devDef[0],devFields[0],devParam,WIDTH,HEIGHT);

  // ---------- Bruitage de l'image déformée ---------
  for(int i = 0; i < WIDTH*HEIGHT ; i++)
  { 
    orig[i] = (float)rand()/RAND_MAX*10.f-5.f;
  }
  hipMemcpy(devOut,orig,taille,hipMemcpyHostToDevice); // Pour ajouter le bruit...
  addVec<<<WIDTH*HEIGHT/1024,1024>>>(devDef[0],devOut); // ..directement sur le device

  // ---------- Pour lire l'image déformée plutôt que la générer -----------
  /*
  readFile("img_d.png",orig, 256);
  hipMemcpy(devDef[0],orig,IMG_SIZE*sizeof(float),hipMemcpyHostToDevice);
  */

  // ---------- Rééchantillonage de l'image pour les différents étages ----------
  gettimeofday(&t1, NULL);
  div = 2;
  for(int i = 1; i < LVL; i++)
  {
    resample<<<gridsize[i],blocksize[i]>>>(devDef[i],devDef[i-1],WIDTH/div);
    div *= 2;
  }
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Rééchantillonage de l'image déformée: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'image déformée ----------
  hipMemcpy(orig,devDef[0],IMG_SIZE*sizeof(float),hipMemcpyDeviceToHost);
  cout << "Image déformée:\n" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

/*
  // ---------- [Facultatif] ecriture en .png des images déformées mippées ----------
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    hipMemcpy(orig,devDef[i],IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
    sprintf(oAddr,"out/mip_%d.png",i);
    writeFile(oAddr, orig, 0, WIDTH/div,HEIGHT/div);
    div *= 2;
  }
*/

  // ---------- Calcul de la Hessienne ----------
  gettimeofday(&t1,NULL);
  //makeMatrix<<<1,tailleMat>>>(devMatrix,texG[0]); // On ne peut pas passer un tableau d'array au device !
  makeHessian(devMatrix,texG[0]);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Génération de la hessienne: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de la Hessienne ----------
  //*
  float test[PARAMETERS*PARAMETERS];
  hipMemcpy(test,devMatrix,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nHessienne:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);
  //*/

  // ---------- Inversion de la hessienne ----------
  gettimeofday(&t1,NULL);
  invert(devMatrix,devInv);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Inversion de la hessienne: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'inverse ----------
  //*
  hipMemcpy(test,devInv,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nMatrice inversée:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);
  //*/

  // ---------- Écriture des paramètres initiaux ----------
  for(int i = 0; i < PARAMETERS; i++)
  param[i] = 0;
  //readParam(argv,param); // Pour tester des valeurs de paramètres par défaut sans recompiler
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  
  div /= 2; // On se sert de la dernière valeur de div (cela equivaut à div = pow(LVL-1,2) )
  // ---------- La boucle principale ---------
  for(int l = LVL-1; l >= 0; l--) // Boucler sur les étages de la pyramide
  {
    cout << " ###  Niveau n°" << l << " ###\n" << endl;
    cout << " Taille de l'image: " << WIDTH/div << "x" << HEIGHT/div << endl;
    gettimeofday(&t00,NULL);
    res = 10000000000; // -- On remet une valeur hénaurme pour être sûr d'avoir une décroissante à la première itération

    for(int i = 0;i < nbIter; i++) // Itérer sur cet étage (en pratique, on fait rarement toutes les itérations)
    {
      // ---------- Infos -------
      gettimeofday(&t0,NULL);
      cout << "Boucle n°" << i+1 << endl;
      hipMemcpy(param,devParam,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Paramètres réels: ";
      for(int j = 0; j < PARAMETERS;j++){cout << paramI[j] << ", ";}
      cout << endl;
      cout << "Paramètres calculés: ";
      for(int j = 0; j < PARAMETERS;j++){cout << param[j] << ", ";}
      cout << endl;
      cout << "Différence: ";
      for(int j = 0; j < PARAMETERS;j++){cout << param[j]-paramI[j] << ", ";}
      cout << endl;

      // --------- Interpolation ----------
      gettimeofday(&t1,NULL);
      deform2D<<<gridsize[l],blocksize[l]>>>(tex[l], devOut, devFields[l], devParam,WIDTH/div,HEIGHT/div); //--
      hipDeviceSynchronize();
      gettimeofday(&t2, NULL);
      cout << "\nInterpolation: " << timeDiff(t1,t2) << "ms." << endl;

/*
      // --------- [Facultatif] Pour enregistrer en .png l'image à chaque itération ----------
      hipMemcpy(orig,devOut,IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
      sprintf(oAddr,"out/devOut%d-%d.png",LVL-l,i);
      writeFile(oAddr,orig,0,WIDTH/div,HEIGHT/div);
*/

/*
      // --------- [Facultatif] Pour enregistrer en .png la différence de l'image ----------
      float def[WIDTH*HEIGHT];
      if(i == 0)
      {
        hipMemcpy(def,devDef[l],IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
      }
      hipMemcpy(orig,devOut,IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
      sprintf(oAddr,"out/diffDevOut%d-%d.png",LVL-l,i);
      writeDiffFile(oAddr,orig,def,4.f,WIDTH/div,HEIGHT/div);
*/

      // ------------ Calcul de la direction de recherche ------------
      gettimeofday(&t1,NULL);
      gradientDescent(texG[l], devOut, devDef[l], devVec, devParam, devFields[l], WIDTH/div, HEIGHT/div); //--
      hipDeviceSynchronize();
      gettimeofday(&t2,NULL);

      // ----------- Affiche le gradient et le temps de calcul -------------
      cout << "Calcul des gradients des paramètres: " << timeDiff(t1,t2) << " ms." << endl;
      hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Gradient des paramètres:" << endl;
      printMat(vec,PARAMETERS,1);
      
      // ---------- Methode de Newton (la matrice est déjà inversée) -------------
      gettimeofday(&t1,NULL);
      myDot<<<1,PARAMETERS,PARAMETERS*sizeof(float)>>>(devInv,devVec,devVec); //--
      //scalMul<<<1,PARAMETERS>>>(devVec,2.f); // Pour un pas fixe
      hipDeviceSynchronize();
      gettimeofday(&t2,NULL);

      // ----------- Affiche le vecteur que l'on va ajouter aux paramètres -----------
      cout << "Mise à jour des valeurs: " << timeDiff(t1,t2) << " ms." << endl;
      hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Direction:" << endl;
      printMat(vec,PARAMETERS,1);

      // ------------ Ajouter tant que la fonctionnelle diminue ---------
      c = 0; // --
      gettimeofday(&t1,NULL);
      while(c<10)
      {
        vecCpy<<<1,PARAMETERS>>>(devVecOld,devParam); //--
        scalMul<<<1,PARAMETERS>>>(devVec,1.f+.15f*c); // -- En augmentant sa taille à chaque fois pour accélérer la convergence
        addVec<<<1,PARAMETERS>>>(devParam,devVec); // --
        deform2D<<<gridsize[l],blocksize[l]>>>(tex[l], devOut, devFields[l], devParam,WIDTH/div,HEIGHT/div); //--
        oldres = res; // --
        res = residuals(devOut, devDef[l], IMG_SIZE/div/div)/IMG_SIZE*div*div; //--
        c++; // -- (quelle ironie...)
        cout << "Ajout: " << c << endl;
        cout << "Résidu: "<< res <<  endl << endl;
        if(res >= oldres) // --
        {
          gettimeofday(&t2,NULL);
          cout << res << " >= " << oldres << "! On annule" << endl;
          vecCpy<<<1,PARAMETERS>>>(devParam,devVecOld); // --
          res = oldres; // --
          cout << c << " ajouts successifs: " << timeDiff(t1,t2) << " ms." << endl;
          cout << "Exécution de toute la boucle: " << timeDiff(t0,t2) << " ms." << endl;
          break; // --
        }
      }
      err = hipGetLastError(); // --
      if(err != hipSuccess) // --
      {cout << "ERREUR !!\n" << hipGetErrorName(err) << endl;exit(-1);} // --
      if(c<=1) // --
      {
        cout << "On n'avance plus... Étage suivant !" << endl;
        gettimeofday(&t2,NULL);
        break; // --
      }
    }
    cout << "Exécution de tout l'étage: " << timeDiff(t00,t2) << " ms." << endl;
    div /= 2; // --
  }

  // ---------- Vérification d'erreur éventuelle ----------
  err = hipGetLastError();
  cout << "Cuda status: " << ((err == 0)?"OK.":"ERREUR !!") << endl;
  cout << err << endl;
  if(err != 0)
  {cout << hipGetErrorName(err) << endl;}

  // ---------- Libération de ce qui a été alloué avec initCuda ----------
  cleanCuda();

  // ---------- Libération des arrays dans la mémoire du device ----------
  hipFree(devOrig);
  hipFree(devGradX);
  hipFree(devGradY);
  hipFree(devParam);
  for(uint i = 0; i < LVL; i++)
  {
    hipFree(devDef[i]);
    hipFree(devFields[i]);
    //hipFree(devG[i]);
  }
  hipFree(devOut);
  hipFree(devMatrix);
  hipFree(devInv);
  hipFree(devVec);
  hipFree(devVecStep);
  hipFree(devVecOld);

  // ---------- Libération des arrays de l'hôte ----------
  free(orig);
  return 0;
}
