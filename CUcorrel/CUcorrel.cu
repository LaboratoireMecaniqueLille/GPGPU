#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "kernels.h"
#include "CUcorrel.h"
#include "util.h"


using namespace std;

int main(int argc, char** argv)
{

  struct timeval t1, t2;
  size_t taille = WIDTH*HEIGHT*sizeof(float);
  size_t taille2 = WIDTH*HEIGHT*sizeof(float2);
  int nbIter=20;
  char iAddr[10] = "img.csv";
  char oAddr[10] = "out0.csv";
  srand(time(NULL));
  float *orig = (float*)malloc(taille);
  for(int i = 0; i < HEIGHT*WIDTH; i++)
  {
    orig[i] = (float)i/WIDTH/HEIGHT+(float)rand()/RAND_MAX/1000;
    //orig[i] = (float)rand()/RAND_MAX;
  }
  float step = 1;
  float vecStep[PARAMETERS] = {0.1,0.1,0.1,1,1,1,1};

  readFile(iAddr,orig,256);
  cout << "Image d'origine" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  dim3 blocksize(min(32,WIDTH),min(32,HEIGHT));
  dim3 gridsize((WIDTH+31)/32,(HEIGHT+31)/32);

  float *devOrig; // Image originale
  float *devDef; //Image déformée à recaler (ici calculée à partir de l'image d'origine)
  float *devGradX; //Gradient de l'image d'origine par rapport à X
  float *devGradY; //.. à Y
  float2 *devFields; // Contient les PARAMETERS champs de déplacements élémentaires dont on cherche l'influence par autant de paramètres
  float *devG; //Les PARAMETERS matrices gradient*champ
  float *devOut; // L'image interpolée à chaque itération


  hipMalloc(&devFields,PARAMETERS*taille2); // Les champs de déformations élémentaires correspondants aux différents modes, placés successivements dans un tableau

  //Assignation des champs
  float2 *field = (float2*)malloc(PARAMETERS*taille2);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 1; // Move X
      field[i+WIDTH*j].y = 0;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Move Y
      field[i+WIDTH*j].y = 1;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 1.4142135624*(j-HEIGHT/2.)/HEIGHT; // Rotation
      field[i+WIDTH*j].y = 1.4142135624*(WIDTH/2.-i)/WIDTH;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+2*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = (float)i/WIDTH; // Stretch X
      field[i+WIDTH*j].y = 0;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+3*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);
  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Stretch Y
      field[i+WIDTH*j].y = (float)j/HEIGHT;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+4*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = (float)j/HEIGHT; // Shear X
      field[i+WIDTH*j].y = 0; 
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+5*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);

  for(int i = 0; i < WIDTH; i++)
  {
    for(int j = 0; j < HEIGHT; j++)
    {
      field[i+WIDTH*j].x = 0; // Shear Y
      field[i+WIDTH*j].y = (float)i/WIDTH;
    }
  }
  //printMat2D(field,WIDTH,HEIGHT,256);
  hipMemcpy(devFields+6*WIDTH*HEIGHT,field,taille2,hipMemcpyHostToDevice);



  hipMalloc(&devOrig,taille);
  hipMalloc(&devDef,taille);
  hipMalloc(&devGradX,taille);
  hipMalloc(&devGradY,taille);


  hipMemcpy(devOrig,orig,taille,hipMemcpyHostToDevice);
  initCuda(devOrig);
  gettimeofday(&t1,NULL);
  gradient<<<gridsize,blocksize>>>(devGradX,devGradY);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "\nCalcul des gradients: " << timeDiff(t1,t2) << " ms." << endl;
  
  //-------- Vérification des gradients -------
  cout << "Gradient Y:" << endl;
  hipMemcpy(orig,devGradY,taille,hipMemcpyDeviceToHost);
  printMat(orig,WIDTH,HEIGHT,256);
  //-------------------------------------------

  gettimeofday(&t1,NULL);
  hipMalloc(&devG,PARAMETERS*taille);
  makeG<<<1,PARAMETERS>>>(devG,devFields,devGradX,devGradY);
  hipDeviceSynchronize();
  if(hipGetLastError() == hipErrorOutOfMemory)
  {cout << "Erreur d'allocation (manque de mémoire graphique ?)" << endl;exit(-1);}
  else if(hipGetLastError() != hipSuccess)
  {cout << "Erreur lors de l'allocation." << endl;exit(-1);}
  gettimeofday(&t2,NULL);
  cout << "Calcul des matrices G: " << timeDiff(t1,t2) << " ms." << endl;


  // ------- POUR VISUALISER G -----------

  /*for(int i = 0;i < PARAMETERS;i++)
  {
  hipMemcpy(orig,devG+i*WIDTH*HEIGHT,taille,hipMemcpyDeviceToHost);
  sprintf(oAddr,"%d",i);
  writeFile(oAddr, orig, 1);                             //
  }
  exit(0);*/
  

/* -- Code pour générer la Hessienne, plus utilisé ---

  hipMalloc(&devMatrix,PARAMETERS*PARAMETERS*sizeof(float));
  dim3 tailleMat(PARAMETERS,PARAMETERS);
  gettimeofday(&t1,NULL);
  makeMatrix<<<1,tailleMat>>>(devMatrix,devG);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Génération de la matrice: " << timeDiff(t1,t2) << " ms." << endl;


  float test[PARAMETERS*PARAMETERS];
  hipMemcpy(test,devMatrix,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nMatrice:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);
*/


  //float param[7] = {2.81,-.86,1.36,.145,4.037,.0036,-4.97};
  float param[7] = {0,0,0,.145,4.037,.0036,-4.97};
  //float param[7] = {7.81,-3.86,6.36,3.145,4.037,.0036,-4.97};
  //float param[7] = {0,0,2,0,0,0,0};
  cout << "Paramètres réels: ";
  for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
  cout << endl;
  float* devParam;
  hipMalloc(&devParam,PARAMETERS*sizeof(float));
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);
  

  deform2D<<<gridsize,blocksize>>>(devDef,devFields,devParam); //Calcule l'image à recaler
  
  hipMemcpy(orig,devDef,taille,hipMemcpyDeviceToHost); // Pour récupérer l'image
  writeFile(oAddr, orig, 1);                             //

  
  hipMalloc(&devOut,taille);

    //param[0] = 2.7;param[1] = -0.86;param[2] = 1.6;param[3] = .345;param[4] = 3.7;param[5] = .06;param[6] = -3.97;
  param[0] = 0;param[1] = 0;param[2] = 0;param[3] = 0;param[4] = 0;param[5] = 0;param[6] = 0;
  //readParam(argv,param); // Pour tester des valeurs sans recompiler
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);


  float res = 10000000000;

  float* devVec;
  float oldres=0;
  hipMalloc(&devVec,PARAMETERS*sizeof(float));
  float vec[PARAMETERS];
  for(int i = 0;i < nbIter; i++)
  {
    cout << "Boucle n°" << i+1 << endl;
    cout << "Paramètres calculés: ";
    for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
    cout << endl;

    gettimeofday(&t1,NULL);
    deform2D<<<gridsize,blocksize>>>(devOut, devFields, devParam);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    cout << "\nInterpolation: " << timeDiff(t1,t2) << "ms." << endl;
    //hipMemcpy(orig,devOut,taille,hipMemcpyDeviceToHost);
    //cout << "Image modifiée:" << endl;
    //printMat(orig,WIDTH,HEIGHT,256);

    gettimeofday(&t1,NULL);
    gradientDescent(devG, devOut, devDef, vec);
    gettimeofday(&t2,NULL);
    cout << "Calcul des gradients des paramètres: " << timeDiff(t1,t2) << " ms." << endl;
    cout << "Valeurs:" << endl;
    printMat(vec,PARAMETERS,1);
    for(int i = 0;i < PARAMETERS; i++)
    {
      param[i] -= step*vecStep[i]*vec[i];
    }
    hipMemcpy(devParam,param,PARAMETERS*sizeof(float),hipMemcpyHostToDevice);
    

    gettimeofday(&t1, NULL);
    oldres = res;
    res = residuals(devOut, devDef, HEIGHT*WIDTH)/HEIGHT/WIDTH;
    if(res <= oldres)
    {step *= 1.2;}
    else
    {step *= -.1;cout << "Reduction du pas !" << endl;}
    //hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    cout << "\nÉcart: "<< res << ", Calcul de l'écart: " << timeDiff(t1,t2) << "ms." << endl;
  }
  int err = 0;
  err = hipGetLastError();
  cout << "Cuda status: " << ((err == 0)?"OK.":"ERREUR !!") << endl;
  cout << err << endl;
  cleanCuda();
  hipFree(devOut);
  hipFree(devG);
  hipFree(devOrig);
  hipFree(devDef);
  hipFree(devGradX);
  hipFree(devGradY);

  return 0;
}
