#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CUcorrel.h"
#include "util.h"

using namespace std;


int main(int argc, char** argv)
{

  struct timeval t0, t1, t2; // Pour mesurer les durées d'exécution
  size_t taille = IMG_SIZE*sizeof(float); // Taille d'un tableau contenant une image
  size_t taille2 = IMG_SIZE*sizeof(float2); // idem à 2 dimensions (fields)
  int nbIter=5; // Le nombre d'itérations
  char iAddr[10] = "img.csv"; // Le nom du fichier à ouvrir
  float *orig = (float*)malloc(taille); // le tableau contenant l'image sur l'hôte
  dim3 blocksize[LVL]; // Pour l'appel aux kernels sur toute l'image (une pour chaque étage)
  dim3 gridsize[LVL];
  uint div = 1; // Pour diviser la taille dans les boucles
  for(int i = 0; i < LVL; i++)
  {
    blocksize[i].x = min(32,WIDTH/div);
    blocksize[i].y = min(32,HEIGHT/div);
    gridsize[i].x = (WIDTH/div+31)/32;
    gridsize[i].y = (HEIGHT/div+31)/32;
    div *= 2;
  }
  dim3 tailleMat(PARAMETERS,PARAMETERS); // La taille de la hessienne

  float *devOrig; // Image originale
  float *devGradX; // Gradient de l'image d'origine par rapport à X
  float *devGradY; // .. à Y
  float2 *devFields[LVL]; // Contient les PARAMETERS champs de déplacements élémentaires à la suite dont on cherche l'influence par autant de paramètres
  float *devG; // Les PARAMETERS matrices gradient*champ
  float *devParam; // Contient la valeur actuelle calculée des paramètres
  float *devDef[LVL]; // Image déformée à recaler (ici calculée à partir de l'image d'origine)
  float *devOut; // L'image interpolée à chaque itération
  float *devMatrix; // La hessienne utilisée pour la méthode de Newton
  float *devInv;  // L'inverse de la Hessienne
  float *devVec; // Vecteur pour stocker les PARAMETERS valeurs du gradient à chaque itération
  float *devVecStep; // Multiplie terme à terme la direction avant le l'ajouter aux paramètres

  srand(time(NULL)); // Seed pour générer le bruit avec rand()

  // ---------- Allocation tous les tableaux du device ---------
  hipMalloc(&devOrig,taille);
  hipMalloc(&devGradX,taille);
  hipMalloc(&devGradY,taille);
  div = 1;
  for(int i = 0; i < LVL; i++)
  {hipMalloc(&devFields[i],PARAMETERS*taille2/div);div*=4;}
  hipMalloc(&devG,PARAMETERS*taille);
  hipMalloc(&devParam,PARAMETERS*sizeof(float));
  div = 1;
  for(int i = 0; i < LVL; i++)
  {hipMalloc(&devDef[i],taille/div);div*=4;}
  hipMalloc(&devOut,taille);
  hipMalloc(&devMatrix,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devInv,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devVec,PARAMETERS*sizeof(float));
  hipMalloc(&devVecStep,PARAMETERS*sizeof(float));
  initCuda();
  if(hipGetLastError() == hipErrorOutOfMemory)
  {cout << "Erreur d'allocation (manque de mémoire graphique ?)" << endl;exit(-1);}
  else if(hipGetLastError() != hipSuccess)
  {cout << "Erreur lors de l'allocation." << endl;exit(-1);}

  // ---------- Lecture du fichier et écriture sur le device ---------
  readFile(iAddr,orig,256);
  hipMemcpy(devOrig,orig,taille,hipMemcpyHostToDevice);

  // ---------- [Facultatif] Affichage de l'image fixe ----------
  cout << "Image d'origine" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  // ---------- Allocation des bindless textures et copie des données ----------
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipArray* cuArray[LVL];
  hipMallocArray(&cuArray[0], &channelDesc,WIDTH,HEIGHT);
  hipMemcpyToArray(cuArray[0],0,0,orig,IMG_SIZE*sizeof(float),hipMemcpyHostToDevice);

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32;
  resDesc.res.linear.sizeInBytes = IMG_SIZE*sizeof(float);
  resDesc.res.linear.devPtr = cuArray[0];

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t tex[LVL]={0};
  hipCreateTextureObject(&tex[0],&resDesc,&texDesc,NULL);

  div = 2;
  for(int i = 1; i < LVL; i++)
  {
  hipMallocArray(&cuArray[i], &channelDesc,WIDTH/div,HEIGHT/div);
  resDesc.res.linear.sizeInBytes = IMG_SIZE/div/div*sizeof(float);
  resDesc.res.linear.devPtr = cuArray[i];
  genMip(tex[i-1],cuArray[i],HEIGHT/div,WIDTH/div);
  hipCreateTextureObject(&tex[i],&resDesc,&texDesc,NULL);
  div *= 2;
  }

  // --------- Calcul des gradients ---------
  gettimeofday(&t1,NULL);
  gradient<<<gridsize[0],blocksize[0]>>>(tex[0],devGradX,devGradY);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "\nCalcul des gradients: " << timeDiff(t1,t2) << " ms." << endl;

  //-------- [Facultatif] Affichage des gradients -------
  /*
  cout << "Gradient X:" << endl;
  hipMemcpy(orig,devGradX,taille,hipMemcpyDeviceToHost);
  printMat(orig,WIDTH,HEIGHT,256);
  */

  // --------- Écriture des fields définis dans fields.cu ----------
  div = 1;
  for(uint i = 0; i < LVL;i++)
  {
    writeFields(devFields[i],WIDTH/div,HEIGHT/div);
    div *= 2;
  }

cout << "OK" << endl;


  // --------- Calcul des matrices G ----------
  gettimeofday(&t1,NULL);
  makeG<<<1,PARAMETERS>>>(devG,devFields[0],devGradX,devGradY);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Calcul des matrices G: " << timeDiff(t1,t2) << " ms." << endl;

  // ------- [Facultatif] Écriture des G en .csv pour les visualiser -----------
  /*
  char oAddr[3];
  for(int i = 0;i < PARAMETERS;i++)
  {
  hipMemcpy(orig,devG+i*WIDTH*HEIGHT,taille,hipMemcpyDeviceToHost);
  sprintf(oAddr,"%d",i);
  writeFile(oAddr, orig, 1);
  }
  */
  
  // --------- Allocation et assignation des paramètres de déformation de devDef ----------
  float param[PARAMETERS] = {-.2,-2.318,3.22,-1.145,1.37,2.3};
  for(int i = 0; i < PARAMETERS; i++)
  {param[i] = 6.f*rand()/RAND_MAX-3.f;}
  cout << "Paramètres réels: ";
  for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
  cout << endl;
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Calcul de l'image à recaler ----------
  deform2D<<<gridsize[0],blocksize[0]>>>(tex[0], devDef[0],devFields[0],devParam,WIDTH,HEIGHT);

  // ---------- Bruitage de l'image déformée ---------
  for(int i = 0; i < WIDTH*HEIGHT ; i++)
  { 
    orig[i] = (float)rand()/RAND_MAX*4-2;
  }
  hipMemcpy(devOut,orig,taille,hipMemcpyHostToDevice);// Pour ajouter le bruit
  addVec<<<WIDTH*HEIGHT/1024,1024>>>(devDef[0],devOut);

  // ---------- Rééchantillonage de l'image pour les différents étages ----------
  div = 2;
  for(int i = 1; i < LVL; i++)
  {
    resample<<<gridsize[i],blocksize[i]>>>(devDef[i],devDef[i-1],WIDTH/div);
    div *= 2;
  }

  // ---------- [Facultatif] Affichage de l'image déformée ----------
  hipMemcpy(orig,devDef[0],IMG_SIZE*sizeof(float),hipMemcpyDeviceToHost);
  cout << "Image déformée:\n" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  // --------- [Facultatif] Écriture de l'image déformée en .csv pour la visualiser ----------
  /*
  char oAddr[10] = "out.csv";
  hipMemcpy(orig,devDef[0],taille,hipMemcpyDeviceToHost); // Pour récupérer l'image
  writeFile(oAddr, orig, 256);
  */

  // ---------- Calcul de la Hessienne ----------
  gettimeofday(&t1,NULL);
  makeMatrix<<<1,tailleMat>>>(devMatrix,devG);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Génération de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de la Hessienne ----------
  float test[PARAMETERS*PARAMETERS];
  hipMemcpy(test,devMatrix,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nHessienne:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);

  // ---------- Inversion de la hessienne ----------
  gettimeofday(&t1,NULL);
  invert(devMatrix,devInv);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Inversion de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'inverse ----------
  hipMemcpy(test,devInv,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nMatrice inversée:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);

  // ---------- Écriture des paramètres initiaux ----------
  for(int i = 0; i < PARAMETERS; i++)
  {param[i] = 0;}
  //readParam(argv,param); // Pour tester des valeurs de paramètres par défaut sans recompiler
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Écriture du pas des paramètres ----------
  float vecStep[PARAMETERS] = {2,2,2,2,2,2};
  hipMemcpy(devVecStep,vecStep,PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  float res = 10000000000; // Le résidu (valeur hénaurme pour être sûr d'avoir une décroissante à la première itération)
  float oldres; // Pour stocker le résidu de l'itération précédente et comparer
  float vec[PARAMETERS]; // Pour stocker sur l'hôte les paramètres calculés
  
  // ---------- La boucle principale ---------
  //Note: seules les instructions marquées par //-- sont réellement nécessaires, les autres sont opour la débug/le timing
  for(int i = 0;i < nbIter; i++)
  {
    gettimeofday(&t0,NULL);
    cout << "Boucle n°" << i+1 << endl;
    hipMemcpy(param,devParam,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Paramètres calculés: ";
    for(int i = 0; i < PARAMETERS;i++){cout << param[i] << ", ";}
    cout << endl;

    gettimeofday(&t1,NULL);
    deform2D<<<gridsize[0],blocksize[0]>>>(tex[0], devOut, devFields[0], devParam,WIDTH,HEIGHT);//--
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    cout << "\nInterpolation: " << timeDiff(t1,t2) << "ms." << endl;

    gettimeofday(&t1,NULL);
    gradientDescent(devG, devOut, devDef[0], devVec);//--
    hipDeviceSynchronize();
    gettimeofday(&t2,NULL);
    cout << "Calcul des gradients des paramètres: " << timeDiff(t1,t2) << " ms." << endl;

    hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Gradient des paramètres:" << endl;
    printMat(vec,PARAMETERS,1);
    
    gettimeofday(&t1,NULL);
    myDot<<<1,PARAMETERS,PARAMETERS*sizeof(float)>>>(devInv,devVec,devVec);//--
    ewMul<<<1,PARAMETERS>>>(devVec,devVecStep);//--
    addVec<<<1,PARAMETERS>>>(devParam,devVec);//--
    hipDeviceSynchronize();
    gettimeofday(&t2,NULL);
    cout << "Mise à jour des valeurs: " << timeDiff(t1,t2) << " ms." << endl;

    hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
    cout << "Direction:" << endl;
    printMat(vec,PARAMETERS,1);

    gettimeofday(&t1, NULL);
    oldres = res;//--
    res = residuals(devOut, devDef[0], IMG_SIZE)/IMG_SIZE;//--
    if(oldres - res < 0)//--
    {cout << "Augmentation de la fonctionnelle !!" << endl;}//--
    gettimeofday(&t2, NULL);
    cout << "\nÉcart: "<< res << ", Calcul de l'écart: " << timeDiff(t1,t2) << "ms." << endl;
    cout << "\nExécution de toute la boucle: " << timeDiff(t0,t2) << "ms.\n**********************\n\n\n" << endl;

  }

  // ---------- Vérification d'erreur éventuelle ----------
  hipError_t err;
  err = hipGetLastError();
  cout << "Cuda status: " << ((err == 0)?"OK.":"ERREUR !!") << endl;
  cout << err << endl;
  if(err != 0)
  {cout << hipGetErrorName(err) << endl;}

  // ---------- Libération de ce qui a été alloué avec initCuda ----------
  cleanCuda();

  // ---------- Libération des arrays dans la mémoire du device ----------
  hipFree(devOrig);
  hipFree(devGradX);
  hipFree(devGradY);
  hipFree(devG);
  hipFree(devParam);
  for(uint i = 0; i < LVL; i++)
  {
    hipFree(devDef[i]);
    hipFree(devFields[i]);
  }
  hipFree(devOut);
  hipFree(devMatrix);
  hipFree(devInv);
  hipFree(devVec);
  hipFree(devVecStep);

  // ---------- Libération des arrays de l'hôte ----------
  free(orig);
  return 0;
}
