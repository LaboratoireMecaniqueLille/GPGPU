#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernels.cuh"
#include "CUcorrel.h"
#include "util.h"

using namespace std;


int main(int argc, char** argv)
{
  struct timeval t0, t1, t2; // Pour mesurer les durées d'exécution
  hipError_t err; // Pour récupérer les erreurs éventuelles
  size_t taille = IMG_SIZE*sizeof(float); // Taille d'un tableau contenant une image
  size_t taille2 = IMG_SIZE*sizeof(float2); // idem à 2 dimensions (fields)
  int nbIter=10; // Le nombre d'itérations
  char iAddr[10] = "img.png"; // Le nom du fichier à ouvrir
  float *orig = (float*)malloc(taille); // le tableau contenant l'image sur l'hôte
  dim3 blocksize[LVL]; // Pour l'appel aux kernels sur toute l'image (une pour chaque étage)
  dim3 gridsize[LVL];
  char oAddr[25]; // pour écrire les noms des fichiers de sortie
  float param[PARAMETERS]; // Stocke les paramètres calculés
  float res; // Le résidu 
  float oldres; // Pour stocker le résidu de l'itération précédente et comparer
  float vec[PARAMETERS]; // Pour stocker sur l'hôte les paramètres calculés
  int c = 0; // Pour compter les boucles et quitter si on ajoute trop
  uint div = 1; // Pour diviser la taille dans les boucles
  for(int i = 0; i < LVL; i++)
  {
    blocksize[i].x = min(32,WIDTH/div);
    blocksize[i].y = min(32,HEIGHT/div);
    blocksize[i].z = 1;
    gridsize[i].x = (WIDTH/div+31)/32;
    gridsize[i].y = (HEIGHT/div+31)/32;
    gridsize[i].z = 1;
    div *= 2;
  }
  dim3 tailleMat(PARAMETERS,PARAMETERS); // La taille de la hessienne

  float *devOrig; // Image originale
  float *devGradX; // Gradient de l'image d'origine par rapport à X
  float *devGradY; // .. à Y
  float2 *devFields[LVL]; // Contient les PARAMETERS champs de déplacements élémentaires à la suite dont on cherche l'influence par autant de paramètres
  float *devG[LVL]; // Les PARAMETERS matrices gradient*champ
  float *devParam; // Contient la valeur actuelle calculée des paramètres
  float *devDef[LVL]; // Image déformée à recaler (ici calculée à partir de l'image d'origine)
  float *devOut; // L'image interpolée à chaque itération
  float *devMatrix; // La hessienne utilisée pour la méthode de Newton
  float *devInv;  // L'inverse de la Hessienne
  float *devVec; // Vecteur pour stocker les PARAMETERS valeurs du gradient à chaque itération
  float *devVecStep; // Multiplie terme à terme la direction avant le l'ajouter aux paramètres
  float *devVecOld; // Pour stocker le vecteur précédent et le restaurer si nécessaire

  srand(time(NULL)); // Seed pour générer le bruit avec rand()

  // ---------- Allocation de tous les tableaux du device ---------
  hipMalloc(&devOrig,taille);
  hipMalloc(&devGradX,taille);
  hipMalloc(&devGradY,taille);
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    hipMalloc(&devFields[i],PARAMETERS*taille2/div);
    hipMalloc(&devG[i],PARAMETERS*taille/div);
    hipMalloc(&devDef[i],taille/div);
    div*=4;
  }
  hipMalloc(&devParam,PARAMETERS*sizeof(float));
  hipMalloc(&devOut,taille);
  hipMalloc(&devMatrix,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devInv,PARAMETERS*PARAMETERS*sizeof(float));
  hipMalloc(&devVec,PARAMETERS*sizeof(float));
  hipMalloc(&devVecStep,PARAMETERS*sizeof(float));
  hipMalloc(&devVecOld,PARAMETERS*sizeof(float));
  initCuda();
  if(hipGetLastError() == hipErrorOutOfMemory)
  {cout << "Erreur d'allocation (manque de mémoire graphique ?)" << endl;exit(-1);}
  else if(hipGetLastError() != hipSuccess)
  {cout << "Erreur lors de l'allocation." << endl;exit(-1);}

  // ---------- Écriture des fields définis dans fields.cu ----------
  div = 1;
  for(uint i = 0; i < LVL;i++)
  {
    writeFields(devFields[i],WIDTH/div,HEIGHT/div);
    div *= 2;
  }

  // ---------- Lecture du fichier et écriture sur le device ---------
  readFile(iAddr,orig,256);
  hipMemcpy(devOrig,orig,taille,hipMemcpyHostToDevice);

  // ---------- [Facultatif] Affichage de l'image fixe ----------
  cout << "Image d'origine" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

  // ---------- Allocation des bindless textures et copie des données ----------
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
  hipArray *cuArray[LVL];
  hipMallocArray(&cuArray[0], &channelDesc,WIDTH,HEIGHT);
  hipMemcpyToArray(cuArray[0],0,0,orig,IMG_SIZE*sizeof(float),hipMemcpyHostToDevice);

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32;
  resDesc.res.linear.sizeInBytes = IMG_SIZE*sizeof(float);
  resDesc.res.linear.devPtr = cuArray[0];

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  texDesc.addressMode[0] = hipAddressModeBorder; //hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeBorder; //hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t tex[LVL]={0};
  hipCreateTextureObject(&tex[0],&resDesc,&texDesc,NULL);

  div = 2;
  for(int i = 1; i < LVL; i++)
  {
    hipMallocArray(&cuArray[i], &channelDesc,WIDTH/div,HEIGHT/div);
    resDesc.res.linear.sizeInBytes = IMG_SIZE/div/div*sizeof(float);
    resDesc.res.linear.devPtr = cuArray[i];
    genMip(tex[i-1],cuArray[i],WIDTH/div,HEIGHT/div);
    hipCreateTextureObject(&tex[i],&resDesc,&texDesc,NULL);
    div *= 2;
  }

  // --------- Calcul des matrices G ----------
  gettimeofday(&t1,NULL);
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    gradient<<<gridsize[i],blocksize[i]>>>(tex[i],devGradX,devGradY, WIDTH/div, HEIGHT/div);
    makeG<<<1,PARAMETERS>>>(devG[i],devFields[i],devGradX,devGradY,WIDTH/div,HEIGHT/div);
    div *= 2;
  }
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Calcul des matrices G: " << timeDiff(t1,t2) << " ms." << endl;

  // ------- [Facultatif] Écriture des G en .csv pour les visualiser -----------
  /*
  div = 1;
  for(int l = 0; l < LVL; l++)
  {
    for(int i = 0;i < PARAMETERS;i++)
    {
    hipMemcpy(orig,devG[l]+i*WIDTH*HEIGHT/div/div,taille/div/div,hipMemcpyDeviceToHost);
    sprintf(oAddr,"out/G%d-%d.png",l,i);
    writeFile(oAddr, orig, 128, WIDTH/div, HEIGHT/div);
    }
    div *= 2;
  }
  */
  
  // --------- Allocation et assignation des paramètres de déformation de devDef ----------
  float paramI[PARAMETERS];
  for(int i = 0; i < PARAMETERS; i++)
  paramI[i] = 80.f*rand()/RAND_MAX-40.f;

  cout << "Paramètres réels: ";
  for(int i = 0; i < PARAMETERS;i++){cout << paramI[i] << ", ";}
  cout << endl;
  hipMemcpy(devParam, paramI, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  // ---------- Calcul de l'image à recaler ----------
  deform2D<<<gridsize[0],blocksize[0]>>>(tex[0], devDef[0],devFields[0],devParam,WIDTH,HEIGHT);

  // ---------- Bruitage de l'image déformée ---------
  for(int i = 0; i < WIDTH*HEIGHT ; i++)
  { 
    orig[i] = (float)rand()/RAND_MAX*4.f-2.f;
  }
  hipMemcpy(devOut,orig,taille,hipMemcpyHostToDevice); // Pour ajouter le bruit...
  addVec<<<WIDTH*HEIGHT/1024,1024>>>(devDef[0],devOut); // ..directement sur le device


  // ---------- Pour lire l'image déformée plutôt que la générer -----------
  /*
  readFile("img_d.png",orig, 256);
  hipMemcpy(devDef[0],orig,IMG_SIZE*sizeof(float),hipMemcpyHostToDevice);
  */

  // ---------- Rééchantillonage de l'image pour les différents étages ----------
  gettimeofday(&t1, NULL);
  div = 2;
  for(int i = 1; i < LVL; i++)
  {
    resample<<<gridsize[i],blocksize[i]>>>(devDef[i],devDef[i-1],WIDTH/div);
    div *= 2;
  }
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Rééchantillonage de l'image déformée: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'image déformée ----------
  hipMemcpy(orig,devDef[0],IMG_SIZE*sizeof(float),hipMemcpyDeviceToHost);
  cout << "Image déformée:\n" << endl;
  printMat(orig,WIDTH,HEIGHT,256);

/*
  // ---------- [Facultatif] ecriture en .png des images déformées mippées ----------
  div = 1;
  for(int i = 0; i < LVL; i++)
  {
    hipMemcpy(orig,devDef[i],IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
    sprintf(oAddr,"out/mip_%d.png",i);
    writeFile(oAddr, orig, 0, WIDTH/div,HEIGHT/div);
    div *= 2;
  }
*/

  // ---------- Calcul de la Hessienne ----------
  gettimeofday(&t1,NULL);
  makeMatrix<<<1,tailleMat>>>(devMatrix,devG[0]);
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);
  cout << "Génération de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de la Hessienne ----------
  /*
  float test[PARAMETERS*PARAMETERS];
  hipMemcpy(test,devMatrix,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nHessienne:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);
  */

  // ---------- Inversion de la hessienne ----------
  gettimeofday(&t1,NULL);
  invert(devMatrix,devInv);
  hipDeviceSynchronize();
  gettimeofday(&t2,NULL);
  cout << "Inversion de la matrice: " << timeDiff(t1,t2) << " ms." << endl;

  // ---------- [Facultatif] Affichage de l'inverse ----------
  /*
  hipMemcpy(test,devInv,PARAMETERS*PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
  cout << "\nMatrice inversée:" << endl;
  printMat(test,PARAMETERS,PARAMETERS);
  */

  // ---------- Écriture des paramètres initiaux ----------
  for(int i = 0; i < PARAMETERS; i++)
  param[i] = 0;
  //readParam(argv,param); // Pour tester des valeurs de paramètres par défaut sans recompiler
  hipMemcpy(devParam, param, PARAMETERS*sizeof(float),hipMemcpyHostToDevice);

  
  div /= 2; // On se sert de la dernière valeur de div (cela equivaut à div = pow(LVL-1,2) )
  // ---------- La boucle principale ---------
  for(int l = LVL-1; l >= 0; l--) // Boucler sur les étages de la pyramide
  {
    cout << " ###  Niveau n°" << l << " ###\n" << endl;
    cout << " Taille de l'image: " << WIDTH/div << "x" << HEIGHT/div << endl;
    res = 10000000000;// On remet une valeur hénaurme pour être sûr d'avoir une décroissante à la première itération

    for(int i = 0;i < nbIter; i++) // Itérer sur cet étage (en pratique, on fait rarement toutes les itérations)
    {
      // ---------- Infos -------
      gettimeofday(&t0,NULL);
      cout << "Boucle n°" << i+1 << endl;
      hipMemcpy(param,devParam,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Paramètres réels: ";
      for(int j = 0; j < PARAMETERS;j++){cout << paramI[j] << ", ";}
      cout << endl;
      cout << "Paramètres calculés: ";
      for(int j = 0; j < PARAMETERS;j++){cout << param[j] << ", ";}
      cout << endl;
      cout << "Différence: ";
      for(int j = 0; j < PARAMETERS;j++){cout << param[j]-paramI[j] << ", ";}
      cout << endl;

      // --------- Interpolation ----------
      gettimeofday(&t1,NULL);
      deform2D<<<gridsize[l],blocksize[l]>>>(tex[l], devOut, devFields[l], devParam,WIDTH/div,HEIGHT/div);//--
      hipDeviceSynchronize();
      gettimeofday(&t2, NULL);
      cout << "\nInterpolation: " << timeDiff(t1,t2) << "ms." << endl;

/*
      // --------- [Facultatif] Pour enregistrer en .png l'image à chaque itération ----------
      hipMemcpy(orig,devOut,IMG_SIZE/div/div*sizeof(float),hipMemcpyDeviceToHost);
      sprintf(oAddr,"out/devOut%d-%d.png",LVL-l,i);
      writeFile(oAddr,orig,1,0, WIDTH/div,HEIGHT/div);
*/
      // ------------ Calcul de la direction de recherche ------------
      gettimeofday(&t1,NULL);
      gradientDescent(devG[l], devOut, devDef[l], devVec, WIDTH/div, HEIGHT/div);//--
      hipDeviceSynchronize();
      gettimeofday(&t2,NULL);

      // ----------- Affiche le gradient et le temps de calcul -------------
      cout << "Calcul des gradients des paramètres: " << timeDiff(t1,t2) << " ms." << endl;
      hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Gradient des paramètres:" << endl;
      printMat(vec,PARAMETERS,1);
      
      // ---------- Methode de Newton (la matrice est déjà inversée) -------------
      gettimeofday(&t1,NULL);
      myDot<<<1,PARAMETERS,PARAMETERS*sizeof(float)>>>(devInv,devVec,devVec);//--
      //scalMul<<<1,PARAMETERS>>>(devVec,2.f); // Pour un pas fixe
      hipDeviceSynchronize();
      gettimeofday(&t2,NULL);

      // ----------- Affiche le vecteur que l'on va ajouter aux paramètres -----------
      cout << "Mise à jour des valeurs: " << timeDiff(t1,t2) << " ms." << endl;
      hipMemcpy(vec,devVec,PARAMETERS*sizeof(float),hipMemcpyDeviceToHost);
      cout << "Direction:" << endl;
      printMat(vec,PARAMETERS,1);

      // ------------ Expérimental: ajouter tant que la fonctionnelle diminue ---------
      c = 0;
      while(c<60)
      {
        vecCpy<<<1,PARAMETERS>>>(devVecOld,devParam);
        scalMul<<<1,PARAMETERS>>>(devVec,1.1f); // En augmentant sa taille à chaque fois pour accélérer la convergence
        addVec<<<1,PARAMETERS>>>(devParam,devVec);
        deform2D<<<gridsize[l],blocksize[l]>>>(tex[l], devOut, devFields[l], devParam,WIDTH/div,HEIGHT/div);//--
        oldres = res;
        res = residuals(devOut, devDef[l], IMG_SIZE/div/div)/IMG_SIZE*div*div;//--
        c++; // <= Haha...
        cout << "Ajout: " << c << endl;
        cout << "Résidu: "<< res <<  endl << endl;
        if(res >= oldres)
        {
          cout << "> " << oldres << "! On annule" << endl;
          vecCpy<<<1,PARAMETERS>>>(devParam,devVecOld);
          res = oldres;
          break;
        }

      }
      err = hipGetLastError();
      if(err != hipSuccess)
      {cout << "ERREUR !!\n" << hipGetErrorName(err) << endl;exit(-1);}
      if(c<=1)
      {
        cout << "On n'avance plus... Boucle suivante !" << endl;
        break;
      }
      
    }
    div /= 2;
  }

  // ---------- Vérification d'erreur éventuelle ----------
  err = hipGetLastError();
  cout << "Cuda status: " << ((err == 0)?"OK.":"ERREUR !!") << endl;
  cout << err << endl;
  if(err != 0)
  {cout << hipGetErrorName(err) << endl;}

  // ---------- Libération de ce qui a été alloué avec initCuda ----------
  cleanCuda();

  // ---------- Libération des arrays dans la mémoire du device ----------
  hipFree(devOrig);
  hipFree(devGradX);
  hipFree(devGradY);
  hipFree(devParam);
  for(uint i = 0; i < LVL; i++)
  {
    hipFree(devDef[i]);
    hipFree(devFields[i]);
    hipFree(devG[i]);
  }
  hipFree(devOut);
  hipFree(devMatrix);
  hipFree(devInv);
  hipFree(devVec);
  hipFree(devVecStep);
  hipFree(devVecOld);

  // ---------- Libération des arrays de l'hôte ----------
  free(orig);
  return 0;
}
